#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void checkPrimeKernel (long long start, long long end, bool *results, long long *numbers) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    long long num = start + (idx * 2); 
    

    if(num > end) return;

    bool isPrime = true;
    

    if (num <= 1) {
        isPrime = false;
        return;
    }

    if (num == 2) {
        isPrime = true;
        return;
    }

    if (num % 2 == 0) {
        isPrime = false;
        return;
    }

    if (num > end) {
        return;
    }

    for (long long i = 3; i * i <= num; i += 2) {
        if(num % i == 0) {
            isPrime = false;
            break;
        }
    }

    numbers[idx] = num;
    results[idx] = isPrime;

}


bool cpuPrimeKernel (long long num) {
    if (num <= 1) return false;
    if (num == 2) return true;
    if (num % 2 == 0) return false;

    for (long long i = 3; i * i <= num; i += 2) {
        if (num % i == 0) {
            return false;
        }
    }

    return true;
}



int main() {
    long long start = 237'023LL;
    long long end = 337'025LL;

    int threadsPerBlock = 256;
    int totalNumbers = (end - start) / 2 + 1 ;
    int blocksPerGrid = (totalNumbers + threadsPerBlock - 1) / threadsPerBlock;
    bool *d_results, *h_results;
    long long *d_numbers, *h_numbers; 

hipMalloc(&d_results, totalNumbers * sizeof(bool));
h_results = new bool[totalNumbers];

hipMalloc(&d_numbers, totalNumbers * sizeof(long long));
h_numbers = new long long[totalNumbers];

    hipEvent_t startEvent, stopEvent;

    hipEventCreate (&startEvent);
    hipEventCreate(&stopEvent);

    hipEventRecord(startEvent, 0);

    checkPrimeKernel<<<blocksPerGrid, threadsPerBlock>>>(start, end, d_results, d_numbers);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    hipMemcpy(h_results, d_results, totalNumbers * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(h_numbers, d_numbers, totalNumbers * sizeof(long long), hipMemcpyDeviceToHost);


    float gpuDuration = 0;
    hipEventElapsedTime(&gpuDuration, startEvent, stopEvent);

    std::cout << "Time taken on GPU: " << gpuDuration << std::endl;
    for (int i = 0; i < totalNumbers; i++){
        if(h_results[i]) {
            std::cout << h_numbers[i] << " ";
        }

        std::cout << std::endl;
    }

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    auto startTime = std::chrono::high_resolution_clock::now();

    for (long long num = start; num <= end; num += 2) {
        cpuPrimeKernel(num);

    auto endTime = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::milli> cpuDuration = endTime - startTime;

    std::cout << "Time on CPU: " << std::fixed << cpuDuration.count() << std::endl;
    std::cout << "speed gain: " << cpuDuration.count() / gpuDuration << std::endl;

    }

    hipFree(d_numbers);
    delete[] h_numbers;



};
