#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <vector>

__global__ void vectorAbsDiff(float *A, float *B, float *C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        C[idx] = sqrtf((A[idx] - B[idx]) * (A[idx] - B[idx]));
    }
}


int main() {

    float h_A[5]= {5.3, 3.4, 2.3, 5.3, 2.4}; 
    float h_B[5]= {3.5, 1.7, 2.5, 5.8, 0.4};
    float h_C[5];
    float *d_A, *d_B, *d_C;
   
    int N = 5;
    int size = N * sizeof(float);

    int threadsPerBlock = 256;
    int Blocks = (N + threadsPerBlock - 1)/ threadsPerBlock;


    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    hipEventRecord(startEvent, 0);

    vectorMul<<<Blocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent); 
    

    float gpuDuration = 0;

    hipEventElapsedTime(&gpuDuration, startEvent, stopEvent);
    
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


    for (int i = 0; i < N; ++i) {
        std::cout << "C[" << i << "] =" << h_C[i] << std::endl;
        };

    std::cout << "Computation performed in " << gpuDuration << "ms." << std::endl;

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}
